#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdlib>
#include <fstream>
#include <ctime>

typedef long long ll;
#define SWAP(x, y)  ll tmp = (x); \
                    (x) = (y); \
                    (y) = (tmp);

__device__ void quickSort(ll* arr, ll low, ll high, ll depth);

// CUDA kernel for partition
__device__ ll partition(ll* arr, ll low, ll high) {
    ll pivot = arr[high];
    ll i = low - 1;

    for (ll j = low; j <= high - 1; ++j) {
        if (arr[j] < pivot) {
            i++;
            SWAP(arr[i], arr[j]);
        }
    }

    SWAP(arr[i + 1], arr[high]);
    return i + 1;
}

__global__ void tap(ll* arr, ll low1, ll high1, ll low2, ll high2, ll depth) {
    int index = threadIdx.x;
    if (index == 0) {
        // printf("[%d]: low1: %10lld, high1: %10lld, depth: %lld\n", index, low1, high1, depth);
        quickSort(arr, low1, high1, depth);  // Left part
    } else if (index == 1) {
        // printf("[%d]: low2: %10lld, high2: %10lld, depth: %lld\n", index, low2, high2, depth);
        quickSort(arr, low2, high2, depth); // Right part
    } else {
        printf("GG\n");
    }
    hipDeviceSynchronize();
}

// CUDA kernel for quicksort
__device__ void quickSort(ll* arr, ll low, ll high, ll depth) {
    if (low < high) {
        ll pi = partition(arr, low, high);

        if (depth > 0) {
            // printf("[C]: low1: %10lld, high1: %10lld, low2: %10lld, high2: %10lld, depth: %d\n", low, pi - 1, pi + 1, high, depth - 1);
            tap<<<1,2>>>(arr, low, pi - 1, pi + 1, high, depth - 1);

        } else {
            quickSort(arr, low, pi - 1, 0);
            quickSort(arr, pi + 1, high, 0);
        }
    } else {
        // printf("[%d] low >= high: %lld >= %lld\n", depth, low, high);
    }
}

__host__ void read_data(std::vector<ll>& arr) {
    std::ifstream inFile("random_numbers.bin", std::ios::binary | std::ios::in);

    if (inFile.is_open()) {
        ll value;
        while (inFile.read(reinterpret_cast<char*>(&value), sizeof(ll))) {
            arr.push_back(value);
        }

        inFile.close();
        std::cout << "Number read from binary file successfully\n";
    } else {
        std::cerr << "Can't open random_numbers.bin\n";
    }
}

__host__ void validate(std::vector<ll>& arr) {
    ll n = arr.size();
    for (ll i = 0; i < n-1; i++) {
        if (arr[i] > arr[i+1]) {
            std::cout << "error at " << i << " " << arr[i] << " " << arr[i+1] << std::endl;
        }
    }
}

int main() {

    std::vector<ll> arr;
    read_data(arr);
    ll N = arr.size();
    std::cout << "Load count:" << N << std::endl;

    // Allocate and copy array to device
    ll* d_arr;
    hipMalloc((void**)&d_arr, N * sizeof(ll));
    hipMemcpy(d_arr, arr.data(), N * sizeof(ll), hipMemcpyHostToDevice);

    // Call CUDA kernel to perform quicksort
    tap<<<1, 1>>>(d_arr, 0, N-1, -1, -1, 2);
    hipDeviceSynchronize();

    // Copy back the sorted array
    hipMemcpy(arr.data(), d_arr, N * sizeof(ll), hipMemcpyDeviceToHost);
    validate(arr);

    hipFree(d_arr);

    return 0;
}
