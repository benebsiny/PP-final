#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdlib>
#include <fstream>
#include <ctime>
#include "commons/helper.hpp"

typedef long long ll;
#define SWAP(x, y) \
    ll tmp = (x);  \
    (x) = (y);     \
    (y) = (tmp);

__device__ void quickSort(ll *arr, ll low, ll high, ll depth);

// CUDA kernel for partition
__device__ ll partition(ll *arr, ll low, ll high)
{
    ll pivot = arr[high];
    ll i = low - 1;

    for (ll j = low; j <= high - 1; ++j)
    {
        if (arr[j] < pivot)
        {
            i++;
            SWAP(arr[i], arr[j]);
        }
    }

    SWAP(arr[i + 1], arr[high]);
    return i + 1;
}

__global__ void tap(ll *arr, ll low1, ll high1, ll low2, ll high2, ll depth)
{
    int index = threadIdx.x;
    if (index == 0)
    {
        quickSort(arr, low1, high1, depth); // Left part
    }
    else if (index == 1)
    {
        quickSort(arr, low2, high2, depth); // Right part
    }
    else
    {
        printf("GG\n");
    }
    hipDeviceSynchronize();
}

// CUDA kernel for quicksort
__device__ void quickSort(ll *arr, ll low, ll high, ll depth)
{
    if (low < high)
    {
        ll pi = partition(arr, low, high);

        if (depth > 0)
        {
            tap<<<1, 2>>>(arr, low, pi - 1, pi + 1, high, depth - 1);
        }
        else
        {
            quickSort(arr, low, pi - 1, 0);
            quickSort(arr, pi + 1, high, 0);
        }
    }
}

__host__ bool read_data(std::vector<long long> &arr, std::string filename)
{
    std::ifstream inFile(filename, std::ios::binary | std::ios::in);

    if (!inFile.is_open())
    {
        return false;
    }

    long long value;
    while (inFile.read(reinterpret_cast<char *>(&value), sizeof(long long)))
    {
        arr.push_back(value);
    }

    inFile.close();

    return true;
}

__host__ void validate(std::vector<ll> &arr)
{
    ll n = arr.size();
    for (ll i = 0; i < n - 1; i++)
    {
        if (arr[i] > arr[i + 1])
        {
            std::cout << "error at " << i << " " << arr[i] << " " << arr[i + 1] << std::endl;
        }
    }
}

int main(int argc, char **argv)
{
    if (argc != 2)
    {
        std::cerr << "[*] Usage: " << argv[0] << " <input file>\n";
        return 1;
    }

    std::vector<ll> arr;
    std::string filename = argv[1];
    if (!read_data(arr, filename))
    {
        std::cerr << "Can't read data\n";
        return 1;
    }
    ll N = arr.size();
    std::cout << "Load count:" << N << std::endl;

    // Allocate and copy array to device
    ll *d_arr;
    hipMalloc((void **)&d_arr, N * sizeof(ll));
    hipMemcpy(d_arr, arr.data(), N * sizeof(ll), hipMemcpyHostToDevice);

    // Call CUDA kernel to perform quicksort
    tap<<<1, 1>>>(d_arr, 0, N - 1, -1, -1, 2);
    hipDeviceSynchronize();

    // Copy back the sorted array
    hipMemcpy(arr.data(), d_arr, N * sizeof(ll), hipMemcpyDeviceToHost);
    validate(arr);

    hipFree(d_arr);

    return 0;
}
